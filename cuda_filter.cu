
#include <hip/hip_runtime.h>
__global__ void invert_kernel(unsigned char* img, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = x * channels;
    if (x < width * height) {
        for (int c = 0; c < channels; ++c) {
            img[idx + c] = 255 - img[idx + c];
        }
    }
}

extern "C"
void cuda_invert(unsigned char* data, int w, int h, int c) {
    int total = w * h;
    unsigned char* d_img;
    hipMalloc(&d_img, total * c);
    hipMemcpy(d_img, data, total * c, hipMemcpyHostToDevice);
    invert_kernel<<<(total + 255) / 256, 256>>>(d_img, w, h, c);
    hipMemcpy(data, d_img, total * c, hipMemcpyDeviceToHost);
    hipFree(d_img);
}
