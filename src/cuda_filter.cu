#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "cuda_filter.h"

// Convert RGB image to grayscale
__global__ void rgb_to_gray_kernel(unsigned char* input, unsigned char* gray, int width, int height, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;
    int i = idx * channels;
    gray[idx] = 0.299f * input[i] + 0.587f * input[i+1] + 0.114f * input[i+2];
}

// Apply Gaussian Blur
__global__ void gaussian_blur_kernel_3x3(unsigned char* gray, unsigned char* blurred, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int sum = 0;
    int i = y * width + x;
    int weights[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int px = x + dx;
            int py = y + dy;
            sum += gray[py * width + px] * weights[dy + 1][dx + 1];
        }
    }
    blurred[i] = sum / 16;
}

// Example 5x5 Gaussian (approx. σ = 1.4, normalized)
__constant__ int gauss5x5[5][5] = {
    {1,  4,  6,  4, 1},
    {4, 16, 24, 16, 4},
    {6, 24, 36, 24, 6},
    {4, 16, 24, 16, 4},
    {1,  4,  6,  4, 1}
};

__global__ void gaussian_blur_kernel_5x5(unsigned char* gray, unsigned char* blurred, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 2 || y < 2 || x >= width - 2 || y >= height - 2) return;

    int sum = 0;
    int weight_sum = 256;
    int i = y * width + x;

    for (int dy = -2; dy <= 2; dy++) {
        for (int dx = -2; dx <= 2; dx++) {
            int px = x + dx;
            int py = y + dy;
            sum += gray[py * width + px] * gauss5x5[dy + 2][dx + 2];
        }
    }
    blurred[i] = sum / weight_sum;
}

// Apply Sobel filter
__global__ void sobel_kernel(unsigned char* blurred, unsigned char* edge, float* direction, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    int Gx = -1 * blurred[(y-1)*width + (x-1)] + 1 * blurred[(y-1)*width + (x+1)]
           -2 * blurred[y*width + (x-1)] + 2 * blurred[y*width + (x+1)]
           -1 * blurred[(y+1)*width + (x-1)] + 1 * blurred[(y+1)*width + (x+1)];

    int Gy = -1 * blurred[(y-1)*width + (x-1)] - 2 * blurred[(y-1)*width + x] - 1 * blurred[(y-1)*width + (x+1)]
           +1 * blurred[(y+1)*width + (x-1)] + 2 * blurred[(y+1)*width + x] + 1 * blurred[(y+1)*width + (x+1)];

    edge[i] = min(255, (int)sqrtf((float)(Gx * Gx + Gy * Gy)));

    float angle = atan2f((float)Gy, (float)Gx) * 180.0f / M_PI;
    direction[i] = angle;

}

// Apply non-maximum suppression
__global__ void non_max_suppression_kernel(unsigned char* gradient, float* direction, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    float angle = direction[i];
    float mag = gradient[i];
    

    float m1 = 0, m2 = 0;

    // Angle normalization: 0°, 45°, 90°, 135°
    angle = fmodf(angle + 180.0f, 180.0f);  // Normalize to [0,180)

    if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle < 180)) {
        m1 = gradient[i + 1];
        m2 = gradient[i - 1];
    } else if (angle >= 22.5 && angle < 67.5) {
        m1 = gradient[(y - 1) * width + (x + 1)];
        m2 = gradient[(y + 1) * width + (x - 1)];
    } else if (angle >= 67.5 && angle < 112.5) {
        m1 = gradient[(y - 1) * width + x];
        m2 = gradient[(y + 1) * width + x];
    } else if (angle >= 112.5 && angle < 157.5) {
        m1 = gradient[(y - 1) * width + (x - 1)];
        m2 = gradient[(y + 1) * width + (x + 1)];
    }

    if (mag >= m1 && mag >= m2) {
        output[i] = (unsigned char)mag;
    } else {
        output[i] = 0;
    }
}

// Apply double thresholding
__global__ void double_threshold_kernel(unsigned char* input, unsigned char* output, int width, int height, unsigned char low_thresh, unsigned char high_thresh) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    unsigned char val = input[idx];
    if (val >= high_thresh) {
        output[idx] = 255;  // Strong edge
    } else if (val >= low_thresh) {
        output[idx] = 100;  // Weak edge
    } else {
        output[idx] = 0;    // Non-edge
    }
}

// Suppress isolated weak edges
__global__ void suppress_weak_clusters_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    if (input[i] == 100) { // weak edge
        int count = 0;
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                if (dx == 0 && dy == 0) continue;
                int ni = (y + dy) * width + (x + dx);
                if (input[ni] == 100 || input[ni] == 255) count++;
            }
        }
        output[i] = (count >= 5) ? 100 : 0;
    } else {
        output[i] = input[i];
    }
}

// DFS-based edge tracking kernel (one pass propagation)
__global__ void edge_tracking_dfs_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    if (input[i] == 255) {
        output[i] = 255;
        return;
    }

    if (input[i] == 100) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                int ni = ny * width + nx;
                if (input[ni] == 255) {
                    output[i] = 255;
                    return;
                }
            }
        }
        output[i] = 0;
    } else {
        output[i] = 0;
    }
}

// Temporal Linking Kernel
__global__ void temporal_link_kernel(unsigned char* curr_edge, unsigned char* prev_edge, unsigned char* output, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    if (curr_edge[idx] && prev_edge[idx])
        output[idx] = 255;  // Reinforce edge
    else
        output[idx] = 0;    // Suppress unstable edge
}

extern "C"
void cuda_canny(unsigned char* input, unsigned char* output, int width, int height, int channels, unsigned char* prev_edge) {
    int img_size = width * height;
    unsigned char *d_input, *d_gray, *d_blur, *d_edge, *d_nms, *d_thresh, *d_final, *d_cleaned, *d_prev_edge, *d_temporal;
    float* d_direction;

    hipMalloc(&d_input, img_size * channels);
    hipMalloc(&d_gray, img_size);
    hipMalloc(&d_blur, img_size);
    hipMalloc(&d_edge, img_size);
    hipMalloc(&d_nms, img_size);
    hipMalloc(&d_thresh, img_size);
    hipMalloc(&d_final, img_size);
    hipMalloc(&d_cleaned, img_size);
    hipMalloc(&d_prev_edge, img_size);
    hipMalloc(&d_temporal, img_size);
    hipMalloc(&d_direction, img_size * sizeof(float));

    hipMemcpy(d_input, input, img_size * channels, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (img_size + threads - 1) / threads;
    rgb_to_gray_kernel<<<blocks, threads>>>(d_input, d_gray, width, height, channels);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15) / 16, (height + 15) / 16);
    gaussian_blur_kernel_5x5<<<numBlocks, threadsPerBlock>>>(d_gray, d_blur, width, height);
    sobel_kernel<<<numBlocks, threadsPerBlock>>>(d_blur, d_edge, d_direction, width, height);
    non_max_suppression_kernel<<<numBlocks, threadsPerBlock>>>(d_edge, d_direction, d_nms, width, height);

    // Apply double thresholding: low = 50, high = 100
    double_threshold_kernel<<<blocks, threads>>>(d_nms, d_thresh, width, height, 50, 100);

    // Suppress weak clusters
    suppress_weak_clusters_kernel<<<numBlocks, threadsPerBlock>>>(d_thresh, d_cleaned, width, height);
    
    // Temporal link kernel
    temporal_link_kernel<<<blocks, threads>>>(d_thresh, d_prev_edge, d_temporal, width, height);

    // Run edge tracking 2 iterations
    edge_tracking_dfs_kernel<<<numBlocks, threadsPerBlock>>>(d_thresh, d_final, width, height);
    edge_tracking_dfs_kernel<<<numBlocks, threadsPerBlock>>>(d_final, d_thresh, width, height);

    hipMemcpy(output, d_thresh, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_blur);
    hipFree(d_edge);
    hipFree(d_nms);
    hipFree(d_thresh);
    hipFree(d_final);
    hipFree(d_cleaned);
    hipFree(d_direction);
}

//  Basic segmentation kernel
__global__ void segment_threshold_kernel(unsigned char* input, unsigned char* mask, int width, int height, int threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    mask[idx] = (input[idx] >= threshold) ? 255 : 0;
}

extern "C"
void cuda_segment(unsigned char* input, unsigned char* output_mask, int w, int h, int c, unsigned char threshold) {
    int img_size = w * h;
    unsigned char *d_input, *d_gray, *d_mask;

    hipMalloc(&d_input, img_size * c);
    hipMalloc(&d_gray, img_size);
    hipMalloc(&d_mask, img_size);

    hipMemcpy(d_input, input, img_size * c, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (img_size + threads - 1) / threads;

    rgb_to_gray_kernel<<<blocks, threads>>>(d_input, d_gray, w, h, c);
    segment_threshold_kernel<<<blocks, threads>>>(d_gray, d_mask, w, h, threshold);

    hipMemcpy(output_mask, d_mask, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_mask);
}