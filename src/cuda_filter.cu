#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "cuda_filter.h"

// Convert RGB image to grayscale
__global__ void rgb_to_gray_kernel(unsigned char* input, unsigned char* gray, int width, int height, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;
    int i = idx * channels;
    gray[idx] = 0.299f * input[i] + 0.587f * input[i+1] + 0.114f * input[i+2];
}

// Apply Gaussian Blur
__global__ void gaussian_blur_kernel(unsigned char* gray, unsigned char* blurred, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int sum = 0;
    int i = y * width + x;
    int weights[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int px = x + dx;
            int py = y + dy;
            sum += gray[py * width + px] * weights[dy + 1][dx + 1];
        }
    }
    blurred[i] = sum / 16;
}
// Apply Sobel filter
__global__ void sobel_kernel(unsigned char* blurred, unsigned char* edge, float* direction, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    int Gx = -1 * blurred[(y-1)*width + (x-1)] + 1 * blurred[(y-1)*width + (x+1)]
           -2 * blurred[y*width + (x-1)] + 2 * blurred[y*width + (x+1)]
           -1 * blurred[(y+1)*width + (x-1)] + 1 * blurred[(y+1)*width + (x+1)];

    int Gy = -1 * blurred[(y-1)*width + (x-1)] - 2 * blurred[(y-1)*width + x] - 1 * blurred[(y-1)*width + (x+1)]
           +1 * blurred[(y+1)*width + (x-1)] + 2 * blurred[(y+1)*width + x] + 1 * blurred[(y+1)*width + (x+1)];

    edge[i] = min(255, (int)sqrtf((float)(Gx * Gx + Gy * Gy)));

    float angle = atan2f((float)Gy, (float)Gx) * 180.0f / M_PI;
    direction[i] = angle;

}

// Apply non-maximum suppression
__global__ void non_max_suppression_kernel(unsigned char* gradient, float* direction, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    float angle = direction[i];
    float mag = gradient[i];
    

    float m1 = 0, m2 = 0;

    // Angle normalization: 0°, 45°, 90°, 135°
    angle = fmodf(angle + 180.0f, 180.0f);  // Normalize to [0,180)

    if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle < 180)) {
        m1 = gradient[i + 1];
        m2 = gradient[i - 1];
    } else if (angle >= 22.5 && angle < 67.5) {
        m1 = gradient[(y - 1) * width + (x + 1)];
        m2 = gradient[(y + 1) * width + (x - 1)];
    } else if (angle >= 67.5 && angle < 112.5) {
        m1 = gradient[(y - 1) * width + x];
        m2 = gradient[(y + 1) * width + x];
    } else if (angle >= 112.5 && angle < 157.5) {
        m1 = gradient[(y - 1) * width + (x - 1)];
        m2 = gradient[(y + 1) * width + (x + 1)];
    }

    if (mag >= m1 && mag >= m2) {
        output[i] = (unsigned char)mag;
    } else {
        output[i] = 0;
    }
}

// Apply double thresholding
__global__ void double_threshold_kernel(unsigned char* input, unsigned char* output, int width, int height, unsigned char low_thresh, unsigned char high_thresh) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    unsigned char val = input[idx];
    if (val >= high_thresh) {
        output[idx] = 255;  // Strong edge
    } else if (val >= low_thresh) {
        output[idx] = 100;  // Weak edge
    } else {
        output[idx] = 0;    // Non-edge
    }
}

// DFS-based edge tracking kernel (one pass propagation)
__global__ void edge_tracking_dfs_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    int i = y * width + x;
    if (input[i] == 255) {
        output[i] = 255;
        return;
    }

    if (input[i] == 100) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                int ni = ny * width + nx;
                if (input[ni] == 255) {
                    output[i] = 255;
                    return;
                }
            }
        }
        output[i] = 0;
    } else {
        output[i] = 0;
    }
}

extern "C"
void cuda_canny(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int img_size = width * height;
    unsigned char *d_input, *d_gray, *d_blur, *d_edge, *d_nms, *d_thresh, *d_final;
    float* d_direction;

    hipMalloc(&d_input, img_size * channels);
    hipMalloc(&d_gray, img_size);
    hipMalloc(&d_blur, img_size);
    hipMalloc(&d_edge, img_size);
    hipMalloc(&d_nms, img_size);
    hipMalloc(&d_thresh, img_size);
    hipMalloc(&d_final, img_size);
    hipMalloc(&d_direction, img_size * sizeof(float));

    hipMemcpy(d_input, input, img_size * channels, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (img_size + threads - 1) / threads;
    rgb_to_gray_kernel<<<blocks, threads>>>(d_input, d_gray, width, height, channels);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15) / 16, (height + 15) / 16);
    gaussian_blur_kernel<<<numBlocks, threadsPerBlock>>>(d_gray, d_blur, width, height);
    sobel_kernel<<<numBlocks, threadsPerBlock>>>(d_blur, d_edge, d_direction, width, height);
    non_max_suppression_kernel<<<numBlocks, threadsPerBlock>>>(d_edge, d_direction, d_nms, width, height);

    // Apply double thresholding: low = 50, high = 100
    double_threshold_kernel<<<blocks, threads>>>(d_nms, d_thresh, width, height, 50, 100);

    // Run edge tracking 2 iterations
    edge_tracking_dfs_kernel<<<numBlocks, threadsPerBlock>>>(d_thresh, d_final, width, height);
    edge_tracking_dfs_kernel<<<numBlocks, threadsPerBlock>>>(d_final, d_thresh, width, height);

    hipMemcpy(output, d_thresh, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_blur);
    hipFree(d_edge);
    hipFree(d_nms);
    hipFree(d_thresh);
    hipFree(d_final);
    hipFree(d_direction);
}