#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "frame_io.h"
#include "cuda_filter.h"

int main() {
    int w, h, c;
    const char* test_img = "frames/frame_0000.jpg";

    unsigned char* img = load_image(test_img, &w, &h, &c);
    if (!img) {
        fprintf(stderr, "Failed to load image: %s\n", test_img);
        return 1;
    }

    unsigned char* mask = (unsigned char*)malloc(w * h);

    // Run segmentation
    cuda_segment(img, mask, w, h, c, 100);  // Simple grayscale threshold

    // Save result
    save_image("test_segment_output.jpg", mask, w, h, 1);
    printf("Segmentation mask saved to test_segment_output.jpg\n");

    free(img);
    free(mask);
    return 0;
}
