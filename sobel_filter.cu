#include <hip/hip_runtime.h>
#include <math.h>

// Convert RGB image to grayscale
__global__ void rgb_to_gray(unsigned char* input, unsigned char* gray, int width, int height, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;
    int i = idx * channels;
    gray[idx] = 0.299f * input[i] + 0.587f * input[i+1] + 0.114f * input[i+2];
}

// Apply Sobel filter
__global__ void sobel_filter(unsigned char* gray, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;
    int i = y * width + x;

    int Gx = 
        -1 * gray[(y-1)*width + (x-1)] + 1 * gray[(y-1)*width + (x+1)] +
        -2 * gray[(y  )*width + (x-1)] + 2 * gray[(y  )*width + (x+1)] +
        -1 * gray[(y+1)*width + (x-1)] + 1 * gray[(y+1)*width + (x+1)];

    int Gy = 
        -1 * gray[(y-1)*width + (x-1)] - 2 * gray[(y-1)*width + (x  )] - 1 * gray[(y-1)*width + (x+1)] +
         1 * gray[(y+1)*width + (x-1)] + 2 * gray[(y+1)*width + (x  )] + 1 * gray[(y+1)*width + (x+1)];

    int mag = min(255, (int)sqrtf(Gx * Gx + Gy * Gy));
    output[i] = (unsigned char)mag;
}

extern "C"
void cuda_sobel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int img_size = width * height;
    unsigned char *d_input, *d_gray, *d_output;

    hipMalloc(&d_input, img_size * channels);
    hipMalloc(&d_gray, img_size);
    hipMalloc(&d_output, img_size);

    hipMemcpy(d_input, input, img_size * channels, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (img_size + threads - 1) / threads;
    rgb_to_gray<<<blocks, threads>>>(d_input, d_gray, width, height, channels);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15)/16, (height + 15)/16);
    sobel_filter<<<numBlocks, threadsPerBlock>>>(d_gray, d_output, width, height);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_output);
}